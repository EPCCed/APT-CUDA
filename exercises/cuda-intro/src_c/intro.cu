/*
 * This is a simple CUDA code that negates an array of integers.
 * It introduces the concepts of device memory management, and
 * kernel invocation.
 *
 * Training material developed by James Perry and Alan Gray
 * Copyright EPCC, The University of Edinburgh, 2010 
 */


#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>

/* Forward Declaration*/
/* Utility function to check for and report CUDA errors */
void checkCUDAError(const char*);

/* The number of integer elements in the array */
#define ARRAY_SIZE 256

/*
 * The number of CUDA blocks and threads per block to use.
 * These should always multiply to give the array size.
 * For the single block kernel, NUM_BLOCKS should be 1 and
 * THREADS_PER_BLOCK should be the array size
 */
#define NUM_BLOCKS  1
#define THREADS_PER_BLOCK 256

/* The actual array negation kernel (basic single block version) */
__global__ void negate(int *d_a)
{
    /* Part 2B: negate an element of d_a */
}

/* Multi-block version of kernel for part 2C */
__global__ void negate_multiblock(int *d_a)
{
    /* Part 2C: negate an element of d_a, using multiple blocks this time */
}

/* Main routine */
int main(int argc, char *argv[])
{
    int *h_a, *h_out;
    int *d_a;

    int i;
    size_t sz = ARRAY_SIZE * sizeof(int);



    /* Print device details */
    int deviceNum;
    hipGetDevice(&deviceNum);
    hipDeviceProp_t prop;
    hipGetDeviceProperties(&prop, deviceNum);
    printf("  Device name: %s\n", prop.name);


    /*
     * allocate memory on host
     * h_a holds the input array, h_out holds the result
     */
    h_a = (int *) malloc(sz);
    h_out = (int *) malloc(sz);

    /*
     * allocate memory on device
     */
    /* Part 1A: allocate device memory */

    /* initialise host arrays */
    for (i = 0; i < ARRAY_SIZE; i++) {
        h_a[i] = i;
        h_out[i] = 0;
    }

    /* copy input array from host to GPU */
    /* Part 1B: copy host array h_a to device array d_a */

    /* run the kernel on the GPU */
    /* Part 2A: configure and launch kernel (un-comment and complete) */
    /* dim3 blocksPerGrid( ); */
    /* dim3 threadsPerBlock( ); */
    /* negate<<< , >>>( ); */

    /* wait for all threads to complete and check for errors */
    hipDeviceSynchronize();
    checkCUDAError("kernel invocation");

    /* copy the result array back to the host */
    /* Part 1C: copy device array d_a to host array h_out */

    checkCUDAError("memcpy");

    /* print out the result */
    printf("Results: ");
    for (i = 0; i < ARRAY_SIZE; i++) {
      printf("%d, ", h_out[i]);
    }
    printf("\n\n");

    /* free device buffer */
    /* Part 1D: free d_a */

    /* free host buffers */
    free(h_a);
    free(h_out);

    return 0;
}


/* Utility function to check for and report CUDA errors */
void checkCUDAError(const char *msg)
{
    hipError_t err = hipGetLastError();
    if( hipSuccess != err) 
    {
        fprintf(stderr, "Cuda error: %s: %s.\n", msg, hipGetErrorString( err) );
        exit(EXIT_FAILURE);
    }                         
}
