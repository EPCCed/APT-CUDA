/* -*- mode: C++; -*- */
/*
 * This is a CUDA code that performs an iterative reverse edge 
 * detection algorithm.
 *
 * Training material developed by James Perry and Alan Gray
 * Copyright EPCC, The University of Edinburgh, 2013 
 */


#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <math.h>

#include <sys/types.h>
#include <sys/time.h>


/* Utility Functions */

/*
 * Function to get an accurate time reading
 */
double get_current_time()
{
   static int start = 0, startu = 0;
   struct timeval tval;
   double result;

   if (gettimeofday(&tval, NULL) == -1)
      result = -1.0;
   else if(!start) {
      start = tval.tv_sec;
      startu = tval.tv_usec;
      result = 0.0;
   }
   else
      result = (double) (tval.tv_sec - start) + 1.0e-6*(tval.tv_usec - startu);

   return result;
}


/* Read the input file containing the edge data */
void datread(char *filename, void *vx, int nx, int ny)
{ 
  FILE *fp;

  int nxt, nyt, i, j, t;

  float *x = (float *) vx;

  if (NULL == (fp = fopen(filename,"r")))
  {
    fprintf(stderr, "datread: cannot open <%s>\n", filename);
    exit(-1);
  }

  fscanf(fp,"%d %d",&nxt,&nyt);

  if (nx != nxt || ny != nyt)
  {
    fprintf(stderr,
            "datread: size mismatch, (nx,ny) = (%d,%d) expected (%d,%d)\n",
            nxt, nyt, nx, ny);
    exit(-1);
  }

  for (j=0; j<ny; j++)
  {
    for (i=0; i<nx; i++)
    {
      fscanf(fp,"%d", &t);
      x[(ny-j-1)*nx + i] = t;
    }
  }

  fclose(fp);
}

/* Write the output image as a PGM file */
void pgmwrite(char *filename, void *vx, int nx, int ny)
{
  FILE *fp;

  int i, j, k, grey;

  float xmin, xmax, tmp;
  float thresh = 255.0;

  float *x = (float *) vx;

  if (NULL == (fp = fopen(filename,"w")))
  {
    fprintf(stderr, "pgmwrite: cannot create <%s>\n", filename);
    exit(-1);
  }

  /*
   *  Find the max and min absolute values of the array
   */

  xmin = fabs(x[0]);
  xmax = fabs(x[0]);

  for (i=0; i < nx*ny; i++)
  {
    if (fabs(x[i]) < xmin) xmin = fabs(x[i]);
    if (fabs(x[i]) > xmax) xmax = fabs(x[i]);
  }

  fprintf(fp, "P2\n");
  fprintf(fp, "# Written by pgmwrite\n");
  fprintf(fp, "%d %d\n", nx, ny);
  fprintf(fp, "%d\n", (int) thresh);

  k = 0;

  for (j=ny-1; j >=0 ; j--)
  {
    for (i=0; i < nx; i++)
    {
      /*
       *  Access the value of x[i][j]
       */

      tmp = x[j*nx+i];

      /*
       *  Scale the value appropriately so it lies between 0 and thresh
       */

      if (xmin < 0 || xmax > thresh)
      {
        tmp = (int) ((thresh*((fabs(tmp-xmin))/(xmax-xmin))) + 0.5);
      }
      else
      {
        tmp = (int) (fabs(tmp) + 0.5);
      }

      /*
       *  Increase the contrast by boosting the lower values
       */
     
      grey = (int) (thresh * sqrt(tmp/thresh));

      fprintf(fp, "%3d ", grey);

      if (0 == (k+1)%16) fprintf(fp, "\n");

      k++;
    }
  }

  if (0 != k%16) fprintf(fp, "\n");
  fclose(fp);
}

/* Simple utility function to check for CUDA runtime errors */
void checkCUDAError(const char *msg)
{
    hipError_t err = hipGetLastError();
    if( hipSuccess != err) 
    {
        fprintf(stderr, "Cuda error: %s: %s.\n", msg, hipGetErrorString( err) );
        exit(EXIT_FAILURE);
    }                         
}

